#include "hip/hip_runtime.h"


#include "freelb.h"
#include "freelb.hh"

using T = FLOAT;
using LatSet = D3Q19<T>;

std::size_t N = 10;
std::size_t Ni = 10;
std::size_t Nj = 10;
std::size_t Nk = 10;

// GenericArray
template <typename T>
__any__ void addGenericArrayImp(cudev::GenericArray<T> &a, std::size_t id, T value) {
  a[id] += value;
}
template <typename T>
__global__ void addGenericArray_kernel(cudev::GenericArray<T> *a, T value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < a->size()) {
    addGenericArrayImp(*a, idx, value);
  }
}
template <typename T>
void addGenericArray(GenericArray<T> &a, T value) {
  const unsigned int blockSize = 32;
  const unsigned int blockNum = (a.size() + blockSize - 1) / blockSize;
  addGenericArray_kernel<<<blockNum, blockSize>>>(a.get_devObj(), value);
}

// Data
template <typename T, typename Base>
void addData(Data<T, Base> &a, T value) {
  addData_kernel<<<1, 1>>>(a.get_devObj(), value);
}
template <typename T, typename Base>
__global__ void addData_kernel(cudev::Data<T, Base> *a, T value) {
  addDataImp(*a, value);
}
template <typename T, typename Base>
__any__ void addDataImp(cudev::Data<T, Base> &a, T value) {
  a.get() += value;
}

// Genericvector
template <typename T>
__any__ void addvectorImp(cudev::Genericvector<T> &a, std::size_t id, T value) {
  a[id] += value;
}
template <typename T>
__global__ void addvector_kernel(cudev::Genericvector<T> *a, T value) {
  std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < a->size()) {
    addvectorImp(*a, idx, value);
  }
}
template <typename T>
void addvector(Genericvector<T> &a, T value) {
  const unsigned int blockSize = 32;
  const unsigned int blockNum = (a.size() + blockSize - 1) / blockSize;
  addvector_kernel<<<blockNum, blockSize>>>(a.get_devObj(), value);
}

// StreamArray
template <typename T>
__any__ void addStreamArrayImp(cudev::StreamArray<T> &a, std::size_t id) {
  a[id] = id + 1;
}
template <typename T>
__global__ void addStreamArray_kernel(cudev::StreamArray<T> *a) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < a->size()) {
    addStreamArrayImp(*a, idx);
  }
}
template <typename T>
void addStreamArray(StreamArray<T> &a) {
  const unsigned int blockSize = 32;
  const unsigned int blockNum = (a.size() + blockSize - 1) / blockSize;
  addStreamArray_kernel<<<blockNum, blockSize>>>(a.get_devObj());
}

void set(StreamArray<T> &arr) {
  for(int i = 0; i < arr.size(); i++) {
    arr[i] = i;
  } 
}
// void print(StreamArray<T> &arr) {
//   for(int i = 0; i < arr.size(); i++) {
//     std::cout << arr[i] << " ";
//   }
//   std::cout << std::endl;
// }

// GenericField
template <typename ArrayType, typename T, typename Base>
__any__ void addGenericFieldImp(cudev::GenericField<ArrayType, Base> &a, std::size_t id, T value) {
  a.get(id) += value;
}
template <typename ArrayType, typename T, typename Base>
__global__ void addGenericField_kernel(cudev::GenericField<ArrayType, Base> *a, T value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < a->getField().size()) {
    addGenericFieldImp(*a, idx, value);
  }
}
template <typename ArrayType, typename T, typename Base>
void addGenericField(GenericField<ArrayType, Base> &a, T value) {
  const unsigned int blockSize = 32;
  const unsigned int blockNum = (a.getField().size() + blockSize - 1) / blockSize;
  addGenericField_kernel<<<blockNum, blockSize>>>(a.get_devObj(), value);
}

// blocklattice
template <typename T, typename LatSet, typename TypePack>
__any__ void addBlockLatticeImp(cudev::BlockLattice<T, LatSet, TypePack> &a, std::size_t id, T value) {
  auto& f = a.template getField<cudev::RHO<T>>();
  for (int i = 0; i < LatSet::q; ++i)
  f.get(id) += value * (LatSet::c[i]*LatSet::c[i]);
}
template <typename T, typename LatSet, typename TypePack>
__global__ void addBlockLattice_kernel(cudev::BlockLattice<T, LatSet, TypePack> *a, T value, std::size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    addBlockLatticeImp(*a, idx, value);
  }
}
template <typename T, typename LatSet, typename TypePack>
void addBlockLattice(BlockLattice<T, LatSet, TypePack> &a, T value) {
  const unsigned int blockSize = 32;
  const unsigned int blockNum = (a.getN() + blockSize - 1) / blockSize;
  addBlockLattice_kernel<<<blockNum, blockSize>>>(a.get_devObj(), value, a.getN());
}

int main() {
  // std::cout << "sizeof(T) = " << sizeof(T) << std::endl;
  // std::cout << "sizeof(Vector<T, 3>) = " << sizeof(Vector<T, 3>) << std::endl;
  BaseConverter<T> BaseConv(LatSet::cs2);
  BaseConv.SimplifiedConverterFromRT(Ni, T{1.}, T{1.});
  AABB<T, 3> cavity(Vector<T, 3>{}, Vector<T, 3>(T(Ni), T(Nj), T(Nk)));
  BlockGeometry3D<T> Geo(Ni, Nj, Nk, 1, cavity, 1);
  // using FIELDS = TypePack<RHO<T>, VELOCITY<T, LatSet::d>, POP<T, LatSet::q>>;
  // ValuePack InitValues(T{1.}, Vector<T, 3>{}, T{});
  using FIELDS = TypePack<RHO<T>>;
  ValuePack InitValues(T{1.});
  BlockLatticeManager<T, LatSet, FIELDS> NSLattice(Geo, BaseConv);

  auto& f = NSLattice.getBlockLat(0);
  std::cout << f.template getField<RHO<T>>().get(0) << std::endl;
  addBlockLattice(NSLattice.getBlockLat(0), T{1.});
  f.template getField<RHO<T>>().copyToHost();
  std::cout << f.template getField<RHO<T>>().get(0) << std::endl;

  // BlockFieldManager<VELOCITY<T, 3>, T, 3> vFM(Geo, Vector<T, 3>{T{1.}, T{2.}, T{3.}});
  // auto& v = vFM.getBlockField(0);
  // VELOCITY<T, 3> v(N, Vector<T, 3>{T{1.}, T{2.}, T{3.}});
  // std::cout << v.get(N-1)[0] << " " << v.get(N-1)[1] << " " << v.get(N-1)[2] << std::endl;
  // addGenericField(v, Vector<T, 3>{T{1.}, T{2.}, T{3.}});
  // vFM.copyToHost();
  // std::cout << v.get(N-1)[0] << " " << v.get(N-1)[1] << " " << v.get(N-1)[2] << std::endl;

  // RHO<T> v(N, T{1.});
  // CONSTRHO<T> v(1, T{1.});
  // addData(v, T{1.});
  // v.copyToHost();
  // std::cout << v.get() << std::endl;

  // Genericvector<int> v(N, 1);
  // std::cout << v[N-1] << std::endl;
  // addvector(v, 1);
  // v.copyToHost();
  // std::cout << v[N-1] << std::endl;

  // StreamArray<T> sarr(100000, T{});
  // int offset = int(std::sqrt(100000));
  // sarr.setOffset(offset);
  // set(sarr);
  // sarr.copyToDevice();
  // // print(sarr);
  // addStreamArray(sarr, T{1.});
  // sarr.dev_rotate();
  // sarr.copyToHost();
  // print(sarr);  
  // Timer MainLoopTimer;
  // MainLoopTimer.START_TIMER();
  // for(int i = 0; i < 10000; ++i) {
  //   addStreamArray(sarr);
  //   // sarr.dev_rotate();
  //   sarr.rotate_dev();
  // }
  // MainLoopTimer.END_TIMER();
  // std::cout << MainLoopTimer.GetDurationCount_Only() <<  std::endl;


  return 0;
}

// ----------------------------------------------------------------

// __global__ void childKernel() {
//     printf("Hello from child kernel\n");
// }

// __global__ void parentKernel() {
//     // Launch child kernel from within the parent kernel
//     childKernel<<<2, 1>>>();
//     printf("Hello from parent kernel\n");
// }

// int main() {
//     // Launch the parent kernel
//     parentKernel<<<2, 1>>>();
//     // Wait for the parent kernel to finish
//     hipDeviceSynchronize();
//     return 0;
// }

// ----------------------------------------------------------------

// class vecarray {
//  public:
//   int *vecptr;  // array of pointers pointing to array
//   int dim;      // store length of each array pointed to

//   __device__ __host__ vecarray();  // constructor
//   __device__ __host__ int sum();   // sum up all the elements in the array being
//                                   // pointed to
// };

// vecarray::vecarray() {
//   vecptr = NULL;
//   dim = 0;
// }

// __device__ __host__ int vecarray::sum() {
//   int j = 0, s = 0;
//   for (j = 0; j < dim; j++) s += vecptr[j];
//   return s;
// }

// __global__ void addvecarray(vecarray *v, int *s) { *s = v->sum(); }

// int main() {        // copy *V to device, do sum() and pass back
//   vecarray *dev_v;  // the result by dev_v
//   vecarray v;
//   int a[3] = {1, 2, 3};  // initialize v manually
//   int result = 0;
//   int *dev_result;
//   v.vecptr = a;
//   v.dim = 3;
//   int *vptr;

//   hipMalloc((void **)&dev_v, sizeof(vecarray));
//   hipMemcpy(dev_v, &v, sizeof(vecarray), hipMemcpyHostToDevice);  // copy class
//   object

// 	hipMalloc((void **)&(vptr), v.dim * sizeof(int));
//   // copy arrays
//   hipMemcpy(vptr, v.vecptr, v.dim * sizeof(int), hipMemcpyHostToDevice);

//   hipMemcpy(&(dev_v->vecptr), &vptr, sizeof(int *), hipMemcpyHostToDevice);


//   hipMalloc((void **)&dev_result, sizeof(int));
//   addvecarray<<<1, 1>>>(dev_v, dev_result);

//   hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
//   printf("the result is %d\n", result);
//   return 0;
// }