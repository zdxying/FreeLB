/* This file is part of FreeLB
 *
 * Copyright (C) 2024 Yuan Man
 * E-mail contact: ymmanyuan@outlook.com
 * The most recent progress of FreeLB will be updated at
 * <https://github.com/zdxying/FreeLB>
 *
 * FreeLB is free software: you can redistribute it and/or modify it under the terms of
 * the GNU General Public License as published by the Free Software Foundation, either
 * version 3 of the License, or (at your option) any later version.
 *
 * FreeLB is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY;
 * without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
 * PURPOSE. See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with FreeLB. If
 * not, see <https://www.gnu.org/licenses/>.
 *
 */

// cavblock3d.cpp

// Lid-driven cavity flow 3d
// this is a benchmark for the freeLB

// the top wall is set with a constant velocity,
// while the other walls are set with a no-slip boundary condition
// Bounce-Back-like method is used:
// Bounce-Back-Moving-Wall method for the top wall
// Bounce-Back method for the other walls

// block data structure is used

#include "freelb.h"
#include "freelb.hh"

using T = float;
using LatSet = D2Q9<T>;

/*----------------------------------------------
                Simulation Parameters
-----------------------------------------------*/
int Ni;
int Nj;
T Cell_Len;
T RT;
int Thread_Num;

// physical properties
T rho_ref;    // g/mm^3
T Dyna_Visc;  // Pa·s Dynamic viscosity of the liquid
T Kine_Visc;  // mm^2/s kinematic viscosity of the liquid
// init conditions
Vector<T, LatSet::d> U_Ini;  // mm/s
T U_Max;

// bcs
Vector<T, LatSet::d> U_Wall;  // mm/s

// Simulation settings
int MaxStep;
int OutputStep;
T tol;
std::string work_dir;

void readParam() {
  iniReader param_reader("cavity2d.ini");
  // mesh
  work_dir = param_reader.getValue<std::string>("workdir", "workdir_");
  // parallel
  Thread_Num = param_reader.getValue<int>("parallel", "thread_num");

  Ni = param_reader.getValue<int>("Mesh", "Ni");
  Nj = param_reader.getValue<int>("Mesh", "Nj");
  Cell_Len = param_reader.getValue<T>("Mesh", "Cell_Len");
  // physical properties
  rho_ref = param_reader.getValue<T>("Physical_Property", "rho_ref");
  Dyna_Visc = param_reader.getValue<T>("Physical_Property", "Dyna_Visc");
  Kine_Visc = param_reader.getValue<T>("Physical_Property", "Kine_Visc");
  // init conditions
  U_Ini[0] = param_reader.getValue<T>("Init_Conditions", "U_Ini0");
  U_Ini[1] = param_reader.getValue<T>("Init_Conditions", "U_Ini1");
  U_Max = param_reader.getValue<T>("Init_Conditions", "U_Max");
  // bcs
  U_Wall[0] = param_reader.getValue<T>("Boundary_Conditions", "Velo_Wall0");
  U_Wall[1] = param_reader.getValue<T>("Boundary_Conditions", "Velo_Wall1");
  // LB
  RT = param_reader.getValue<T>("LB", "RT");
  // Simulation settings
  MaxStep = param_reader.getValue<int>("Simulation_Settings", "TotalStep");
  OutputStep = param_reader.getValue<int>("Simulation_Settings", "OutputStep");
  tol = param_reader.getValue<T>("tolerance", "tol");


  std::cout << "------------Simulation Parameters:-------------\n" << std::endl;
  std::cout << "[Simulation_Settings]:" << "TotalStep:         " << MaxStep << "\n"
            << "OutputStep:        " << OutputStep << "\n"
            << "Tolerance:         " << tol << "\n"
            << "----------------------------------------------" << std::endl;
}

int main() {
  constexpr std::uint8_t VoidFlag = std::uint8_t(1);
  constexpr std::uint8_t AABBFlag = std::uint8_t(2);
  constexpr std::uint8_t BouncebackFlag = std::uint8_t(4);
  constexpr std::uint8_t BBMovingWallFlag = std::uint8_t(8);

  Printer::Print_BigBanner(std::string("Initializing..."));

  readParam();

  // converters
  BaseConverter<T> BaseConv(LatSet::cs2);
  BaseConv.ConvertFromRT(Cell_Len, RT, rho_ref, Ni * Cell_Len, U_Max, Kine_Visc);
  UnitConvManager<T> ConvManager(&BaseConv);
  ConvManager.Check_and_Print();

  // ------------------ define geometry ------------------
  AABB<T, 2> cavity(Vector<T, 2>(T(0), T(0)),
                    Vector<T, 2>(T(Ni * Cell_Len), T(Nj * Cell_Len)));
  AABB<T, 2> toplid(Vector<T, 2>(Cell_Len, T((Nj - 1) * Cell_Len)),
                    Vector<T, 2>(T((Ni - 1) * Cell_Len), T(Nj * Cell_Len)));
  BlockGeometry2D<T> Geo(Ni, Nj, 1, cavity, Cell_Len);

  // ------------------ define flag field ------------------
  BlockFieldManager<FLAG, T, LatSet::d> FlagFM(Geo, VoidFlag);
  FlagFM.forEach(cavity,
                 [&](FLAG& field, std::size_t id) { field.SetField(id, AABBFlag); });
  FlagFM.template SetupBoundary<LatSet>(cavity, BouncebackFlag);
  FlagFM.forEach(toplid, [&](FLAG& field, std::size_t id) {
    if (util::isFlag(field.get(id), BouncebackFlag)) field.SetField(id, BBMovingWallFlag);
  });
  // do not forget to copy to device
  FlagFM.copyToDevice();

  // vtmwriter::ScalarWriter FlagWriter("flag", FlagFM);
  // vtmwriter::vtmWriter<T, LatSet::d> GeoWriter("GeoFlag", Geo);
  // GeoWriter.addWriterSet(FlagWriter);
  // GeoWriter.WriteBinary();

  // ------------------ define lattice ------------------
  using FIELDS = TypePack<RHO<T>, VELOCITY<T, LatSet::d>, POP<T, LatSet::q>>;
  using cudevFIELDS = typename ExtractCudevFieldPack<FIELDS>::cudev_pack;

  using HostCell = Cell<T, LatSet, FIELDS>;
  using CELL = cudev::Cell<T, LatSet, cudevFIELDS>;

  ValuePack InitValues(BaseConv.getLatRhoInit(), Vector<T, LatSet::d>{}, T{});
  // lattice
  BlockLatticeManager<T, LatSet, FIELDS> NSLattice(Geo, InitValues, BaseConv);
  NSLattice.EnableToleranceU();
  T res = 1;
  // set initial value of field
  Vector<T, 2> LatU_Wall = BaseConv.getLatticeU(U_Wall);
  NSLattice.getField<VELOCITY<T, LatSet::d>>().forEach(
    toplid, FlagFM, BBMovingWallFlag,
    [&](auto& field, std::size_t id) { field.SetField(id, LatU_Wall); }); 

  // bcs
  // BBLikeFixedBlockBdManager<bounceback::normal<CELL>, BlockLatticeManager<T, LatSet, FIELDS>, BlockFieldManager<FLAG, T, LatSet::d>>
  //   NS_BB("NS_BB", NSLattice, FlagFM, BouncebackFlag, VoidFlag);
  // BBLikeFixedBlockBdManager<bounceback::movingwall<CELL>, BlockLatticeManager<T, LatSet, FIELDS>, BlockFieldManager<FLAG, T, LatSet::d>>
  //   NS_BBMW("NS_BBMW", NSLattice, FlagFM, BBMovingWallFlag, VoidFlag);
  // BlockBoundaryManager BM(&NS_BB, &NS_BBMW);

  // define task/ dynamics:
  // bulk task
  using BulkTask = tmp::Key_TypePair<AABBFlag, collision::BGK<moment::rhoU<CELL>, equilibrium::SecondOrder<CELL>>>;
  // wall task
  using WallTask = tmp::Key_TypePair<BouncebackFlag | BBMovingWallFlag, collision::BGK<moment::useFieldrhoU<CELL>, equilibrium::SecondOrder<CELL>>>;
  // BCs task as a collision process, if used, bcs will be handled in the collision process
  using BBTask = tmp::Key_TypePair<BouncebackFlag, collision::BounceBack<CELL>>;
  using BBMVTask = tmp::Key_TypePair<BBMovingWallFlag, collision::BounceBackMovingWall<CELL>>;
  // task collection
  // using TaskCollection = tmp::TupleWrapper<BulkTask, WallTask>;
  using TaskCollection = tmp::TupleWrapper<BulkTask, BBTask, BBMVTask>;
  // task executor
  using NSTask = tmp::TaskSelector<TaskCollection, std::uint8_t, CELL>;

  // task: update rho and u
  using RhoUTask = tmp::Key_TypePair<AABBFlag, moment::rhoU<CELL, true>>;
  using TaskCollectionRhoU = tmp::TupleWrapper<RhoUTask>;
  using TaskSelectorRhoU = tmp::TaskSelector<TaskCollectionRhoU, std::uint8_t, CELL>;

  // writers
  vtmwriter::ScalarWriter RhoWriter("Rho", NSLattice.getField<RHO<T>>());
  vtmwriter::VectorWriter VecWriter("Velocity", NSLattice.getField<VELOCITY<T, LatSet::d>>());
  vtmwriter::vtmWriter<T, LatSet::d> NSWriter("cavblock2dcu", Geo);
  NSWriter.addWriterSet(RhoWriter, VecWriter);

  Printer::Print_BigBanner(std::string("Start Calculation..."));
  std::cout << "Total Cells: " << Geo.getTotalCellNum() << std::endl;

  NSLattice.getField<POP<T, LatSet::q>>().copyToDevice();
  NSLattice.getField<RHO<T>>().copyToDevice();
  NSLattice.getField<VELOCITY<T, LatSet::d>>().copyToDevice();

  // count and timer
  Timer MainLoopTimer;
  Timer OutputTimer;
  
  NSWriter.WriteBinary(MainLoopTimer());

  Printer::Print_BigBanner(std::string("Start Calculation..."));

  while (MainLoopTimer() < MaxStep && res > tol) {

    // NSLattice.ApplyCellDynamics<NSTask>(FlagFM);
    NSLattice.CuDevApplyCellDynamics<NSTask>(FlagFM);
    // NSLattice.CuDevApplyCellDynamics<collision::BGK<moment::rhoU<CELL>, equilibrium::SecondOrder<CELL>>>();
    
    // NSLattice.Stream();
    NSLattice.CuDevStream();
    // BM.Apply(MainLoopTimer());

    // NSLattice.Communicate(MainLoopTimer());

    ++MainLoopTimer;
    ++OutputTimer;

    if (MainLoopTimer() % OutputStep == 0) {
      NSLattice.CuDevApplyCellDynamics<TaskSelectorRhoU>(FlagFM);
      hipDeviceSynchronize();
      NSLattice.getBlockLat(0).getField<RHO<T>>().copyToHost();
      NSLattice.getBlockLat(0).getField<VELOCITY<T, LatSet::d>>().copyToHost();
      
      res = NSLattice.getToleranceU(-1);
      OutputTimer.Print_InnerLoopPerformance(Geo.getTotalCellNum(), OutputStep);
      Printer::Print_Res<T>(res);
      Printer::Endl();
      NSWriter.WriteBinary(MainLoopTimer());
    }
  }

  Printer::Print_BigBanner(std::string("Calculation Complete!"));
  MainLoopTimer.Print_MainLoopPerformance(Geo.getTotalCellNum());
  Printer::Print("Total PhysTime", BaseConv.getPhysTime(MainLoopTimer()));
  Printer::Endl();

  return 0;
}